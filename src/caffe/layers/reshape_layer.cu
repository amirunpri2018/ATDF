#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/reshape_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ReshapeForward(const int nthreads, const Dtype* bottom_data,
    Dtype* top_data, const int bc, const int bh, const int bw, const int tc,
    const int th, const int tw, const int ps) {
  CUDA_KERNEL_LOOP(idx, nthreads) {
    const int old_n = idx/bc/bh/bw;
    const int old_c = (idx/bh/bw)%bc;
    const int old_h = (idx/bw)%bh;
    const int old_w = idx%bw;
    const int new_c = old_c/(ps*ps);
    const int new_h = old_h*ps + (old_c%(ps*ps))/ps;
    const int new_w = old_w*ps + (old_c%(ps*ps))%ps;
    int top_index = ((old_n*tc + new_c)*th + new_h)*tw + new_w;
    top_data[top_index] = bottom_data[idx];
  }
}

template <typename Dtype>
__global__ void ReshapeBackward(const int nthreads, Dtype* bottom_diff,
    const Dtype* top_diff, const int bc, const int bh, const int bw,
    const int tc, const int th, const int tw, const int ps) {
  CUDA_KERNEL_LOOP(idx, nthreads) {
    const int new_n = idx/tc/th/tw;
    const int new_c = (idx/th/tw)%tc;
    const int new_h = (idx/tw)%th;
    const int new_w = idx%tw;
    const int old_c = new_c*ps*ps + (new_h%ps)*ps + new_w%ps;
    const int old_h = new_h/ps;
    const int old_w = new_w/ps;
    const int bottom_index = ((new_n*bc + old_c)*bh + old_h)*bw + old_w;
    bottom_diff[bottom_index] = top_diff[idx];
  }
}

template <typename Dtype>
void ReshapeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const int ps = this->layer_param_.reshape_param().pixelshuffler();
  if (ps != 0) {
    const int count = bottom[0]->count();
    vector<int> bottom_shape = bottom[0]->shape();
    const int bc = bottom_shape[1];
    const int bh = bottom_shape[2];
    const int bw = bottom_shape[3];
    vector<int> top_shape = top[0]->shape();
    const int tc = top_shape[1];
    const int th = top_shape[2];
    const int tw = top_shape[3];
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    ReshapeForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, top_data, bc, bh, bw, tc, th, tw, ps);
  }
}

template <typename Dtype>
void ReshapeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const int ps = this->layer_param_.reshape_param().pixelshuffler();
  if (ps != 0) {
    const int count = top[0]->count();
    vector<int> top_shape = top[0]->shape();
    const int tc = top_shape[1];
    const int th = top_shape[2];
    const int tw = top_shape[3];
    vector<int> bottom_shape = bottom[0]->shape();
    const int bc = bottom_shape[1];
    const int bh = bottom_shape[2];
    const int bw = bottom_shape[3];
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* top_diff = top[0]->gpu_diff();
    ReshapeBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_diff, top_diff, bc, bh, bw, tc, th, tw, ps);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ReshapeLayer);

}  // namespace caffe
